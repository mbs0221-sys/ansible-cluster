#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int main(void) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    std::cout << "[Vector addition of " << numElements << " elements]\n";

    // Allocate the host input vectors A and B
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL) {
        std::cerr << "Failed to allocate host vectors!\n";
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vectors A and B
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess) {
        std::cerr << "Failed to allocate device vector A (error code " << hipGetErrorString(err) << ")!\n";
        exit(EXIT_FAILURE);
    }

    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess) {
        std::cerr << "Failed to allocate device vector B (error code " << hipGetErrorString(err) << ")!\n";
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess) {
        std::cerr << "Failed to allocate device vector C (error code " << hipGetErrorString(err) << ")!\n";
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in device memory
    std::cout << "Copy input data from the host memory to the CUDA device\n";
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        std::cerr << "Failed to copy vector A from host to device (error code " << hipGetErrorString(err) << ")!\n";
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        std::cerr << "Failed to copy vector B from host to device (error code " << hipGetErrorString(err) << ")!\n";
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    std::cout << "CUDA kernel launch with " << blocksPerGrid << " blocks of " << threadsPerBlock << " threads\n";
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    err = hipGetLastError();

    if (err != hipSuccess) {
        std::cerr << "Failed to launch vectorAdd kernel (error code " << hipGetErrorString(err) << ")!\n";
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector in host memory
    std::cout << "Copy output data from the CUDA device to the host memory\n";
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess) {
        std::cerr << "Failed to copy vector C from device to host (error code " << hipGetErrorString(err) << ")!\n";
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            std::cerr << "Result verification failed at element " << i << "!\n";
            exit(EXIT_FAILURE);
        }
    }

    std::cout << "Test PASSED\n";

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess) {
        std::cerr << "Failed to free device vector A (error code " << hipGetErrorString(err) << ")!\n";
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess) {
        std::cerr << "Failed to free device vector B (error code " << hipGetErrorString(err) << ")!\n";
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess) {
        std::cerr << "Failed to free device vector C (error code " << hipGetErrorString(err) << ")!\n";
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    std::cout << "Done\n";
    return 0;
}
